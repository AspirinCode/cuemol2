
#include <hip/hip_runtime.h>
// -*-Mode: C++;-*-
//
//  CUDA test
//

#define THR_PER_BLK 1024

__global__ void test_kernel(float *input, float *output, int len)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < len)
    output[i] = input[i];
}


void launchTestKernel(float *input, float *output, int nlen)
{

  int ngrd;
  if (nlen%THR_PER_BLK==0)
    ngrd = nlen/THR_PER_BLK;
  else
    ngrd = nlen/THR_PER_BLK + 1;

  // execute the kernel
  dim3 block(THR_PER_BLK, 1, 1);
  dim3 grid(ngrd, 1, 1);
  test_kernel<<<grid, block>>>(input, output, nlen);
}


