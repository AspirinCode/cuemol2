
#include <hip/hip_runtime.h>
// -*-Mode: C++;-*-
//
//  CUDA test
//

#define THR_PER_BLK 1024

__global__ void test_kernel(unsigned char *input, unsigned char *output, int len)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < len)
    output[len-i] = input[i];
}


void launchTestKernel(unsigned char *input, unsigned char *output, int nlen)
{

  int ngrd;
  if (nlen%THR_PER_BLK==0)
    ngrd = nlen/THR_PER_BLK;
  else
    ngrd = nlen/THR_PER_BLK + 1;

  // execute the kernel
  dim3 block(THR_PER_BLK, 1, 1);
  dim3 grid(ngrd, 1, 1);
  test_kernel<<<grid, block>>>(input, output, nlen);
}

/*
#include <common.h>
#include <sysdep/CudartCompContext.hpp>

void launchTestKernel(const gfx::ComputeArray *pCA_in, gfx::ComputeArray *pCA_out)
{
  const sysdep::CudartCompArray *pcin = static_cast<const sysdep::CudartCompArray *>(pCA_in);
  float *input = (float *) pcin->getHandle();
  
  sysdep::CudartCompArray *pcout = static_cast<sysdep::CudartCompArray *>(pCA_out);
  float *output = (float *) pcout->getHandle();
  
  int nlen = pCA_in->getElemCount();

  int ngrd;
  if (nlen%THR_PER_BLK==0)
    ngrd = nlen/THR_PER_BLK;
  else
    ngrd = nlen/THR_PER_BLK + 1;

  // execute the kernel
  dim3 block(THR_PER_BLK, 1, 1);
  dim3 grid(ngrd, 1, 1);
  test_kernel<<<grid, block>>>(input, output, nlen);
}
*/


