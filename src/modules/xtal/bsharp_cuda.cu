#include "hip/hip_runtime.h"
// -*-Mode: C++;-*-
//
//  CUDA bsharp implementation
//

#include "StrFac.hpp"

using namespace xtal;

__global__
void convtoary_herm_kern(StrFac *dp_hkl, int HKLSZ,
                         float *dp_recip, int NX, int NY, int NZ, int NXX,
                         float fscl, float b_factor,
                         float m00, float m11, float m22, float m01, float m02, float m12)
{
  const int ithr = blockIdx.x*blockDim.x + threadIdx.x;
  if (ithr>=HKLSZ) return;
  
  int ih = dp_hkl[ithr].ih;
  int ik = dp_hkl[ithr].ik;
  int il = dp_hkl[ithr].il;

  float f_re = dp_hkl[ithr].f_re;
  float f_im = dp_hkl[ithr].f_im;

  float irs = ih*(ih*m00 + ik*m01 + il*m02) + ik*(ik*m11 + il*m12) + il*(il*m22);
  float bscl = float(exp(-b_factor * irs * 0.25f));

  ih = (ih+10000*NX)%NX;
  ik = (ik+10000*NY)%NY;
  il = (il+10000*NZ)%NZ;

  // Make Friedel pair index
  int mh = (NX-ih)%NX;
  int mk = (NY-ik)%NY;
  int ml = (NZ-il)%NZ;

  float re = fscl * bscl * f_re;
  float im = fscl * bscl * f_im;

  int idx;

  // Hermitian case: fill the hemisphere (of L>ncc)
  //  with the Friedel pairs of the refls.
  if (ih<NXX) {
    idx = ih + (ik + il*NY)*NXX;
    dp_recip[idx*2+0] = re;
    dp_recip[idx*2+1] = -im;
  }
  if (mh<NXX) {
    idx = mh + (mk + ml*NY)*NXX;
    dp_recip[idx*2+0] = re;
    dp_recip[idx*2+1] = im;
  }
}

void convtoary_herm_cuda(int NBLK, int NTHR,
                         StrFac *dp_hkl, int HKLSZ,
                         float *dp_recip, int NX, int NY, int NZ, int NXX,
                         float fscl, float b_factor,
                         float m00, float m11, float m22, float m01, float m02, float m12)
{
  convtoary_herm_kern<<<NBLK, NTHR>>>(dp_hkl, HKLSZ,
                                      dp_recip, NX, NY, NZ, NXX,
                                      fscl, b_factor,
                                      m00, m11, m22, m01, m02, m12);
}

//////////

__inline__ __device__
float MIN(float x, float y) {
  return ((x < y) ? x : y);
}

__inline__ __device__
float MAX(float x, float y) {
  return ((x > y) ? x : y);
}

__inline__ __device__
void warpReduce(float &val, float &val2, float &vmin, float &vmax)
{
  val += __shfl_down(val, 16);
  val2 += __shfl_down(val2, 16);
  vmin = MIN(vmin, __shfl_down(vmin, 16));
  vmax = MAX(vmax, __shfl_down(vmax, 16));

  val += __shfl_down(val, 8);
  val2 += __shfl_down(val2, 8);
  vmin = MIN(vmin, __shfl_down(vmin, 8));
  vmax = MAX(vmax, __shfl_down(vmax, 8));

  val += __shfl_down(val, 4);
  val2 += __shfl_down(val2, 4);
  vmin = MIN(vmin, __shfl_down(vmin, 4));
  vmax = MAX(vmax, __shfl_down(vmax, 4));

  val += __shfl_down(val, 2);
  val2 += __shfl_down(val2, 2);
  vmin = MIN(vmin, __shfl_down(vmin, 2));
  vmax = MAX(vmax, __shfl_down(vmax, 2));

  val += __shfl_down(val, 1);
  val2 += __shfl_down(val2, 1);
  vmin = MIN(vmin, __shfl_down(vmin, 1));
  vmax = MAX(vmax, __shfl_down(vmax, 1));
}

__inline__ __device__
void blkReduce(float &val, float &val2, float &vmin, float &vmax)
{
  static __shared__ float shared[32*4]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  warpReduce(val, val2, vmin, vmax);

  // Write reduced value to shared memory
  if (lane==0) {
    shared[wid*4+0] = val;
    shared[wid*4+1] = val2;
    shared[wid*4+2] = vmin;
    shared[wid*4+3] = vmax;
  }

  // Wait for all partial reductions
  __syncthreads();

  //read from shared memory only if that warp existed
  if (threadIdx.x < blockDim.x / warpSize) {
    val = shared[lane*4+0];
    val2 = shared[lane*4+1];
    vmin = shared[lane*4+2];
    vmax = shared[lane*4+3];
  }
  else {
    val = 0.0f;
    val2 = 0.0f;
    vmin = 1.0e10f;
    vmax = -1.0e10f;
  }

  if (wid==0) {
    warpReduce(val, val2, vmin, vmax);
  }
}

__global__
void mapstat4_kern(float *dp_map, int NSIZE, float *dp_tmp)
{
  const int ithr = blockIdx.x*blockDim.x + threadIdx.x;

  float val = (ithr<NSIZE)?dp_map[ithr]:0.0f;
  float val2 = val*val;
  float vmin = val;
  float vmax = val;

  blkReduce(val, val2, vmin, vmax);

  if (threadIdx.x == 0) {
    dp_tmp[blockIdx.x * 4 + 0] = val;
    dp_tmp[blockIdx.x * 4 + 1] = val2;
    dp_tmp[blockIdx.x * 4 + 2] = vmin;
    dp_tmp[blockIdx.x * 4 + 3] = vmax;
  }

  /*
  if (threadIdx.x == 0 && blockIdx.x<5) {
    //printf("mapstat4_kern %d : val=%f, val2=%f, vmin=%f, vmax=%f\n", blockIdx.x, val, val2, vmin, vmax);
    printf("mapstat4_kern %d : val=%f, val2=%f, vmin=%f, vmax=%f\n",blockIdx.x,
           dp_tmp[blockIdx.x * 4 + 0],
           dp_tmp[blockIdx.x * 4 + 1],
           dp_tmp[blockIdx.x * 4 + 2],
           dp_tmp[blockIdx.x * 4 + 3]);
  }
  */
}

void mapstat4_cuda(int NBLK, int NTHR, float *dp_map, int NSIZE, float *dp_tmp)
{
  mapstat4_kern<<<NBLK,NTHR>>>(dp_map, NSIZE, dp_tmp);
}

__global__
void mapstat4_kern2(float *dp_in, int NSIZE)
{
  const int ithr = blockIdx.x*blockDim.x + threadIdx.x;

  float val,val2,vmin,vmax;

  if (ithr<NSIZE) {
    val = dp_in[ithr*4+0];
    val2 = dp_in[ithr*4+1];
    vmin = dp_in[ithr*4+2];
    vmax = dp_in[ithr*4+3];
  }
  else {
    val = 0.0f;
    val2 = 0.0f;
    vmin = 1.0e10f;
    vmax = -1.0e10f;
  }

  blkReduce(val, val2, vmin, vmax);

  if (threadIdx.x == 0) {
    dp_in[blockIdx.x * 4 + 0] = val;
    dp_in[blockIdx.x * 4 + 1] = val2;
    dp_in[blockIdx.x * 4 + 2] = vmin;
    dp_in[blockIdx.x * 4 + 3] = vmax;
  }
}

void mapstat4_cuda2(int NBLK, int NTHR, float *dp_in, int NSIZE)
{
  mapstat4_kern2<<<NBLK,NTHR>>>(dp_in, NSIZE);
}

//////////

__global__
void makebmap_kern(float *dp_map, int NSIZE, float *dp_tmp, unsigned char *dp_bmap)
{
  const int ithr = blockIdx.x*blockDim.x + threadIdx.x;
  if (ithr>=NSIZE) return;

  float rhomin = dp_tmp[2];
  float rhomax = dp_tmp[3];
  float step = (rhomax - rhomin)/256.0f;
  float base = rhomin;

  float rho = dp_map[ithr];
  rho = (rho-base)/step;
  if (rho<0.0f) rho = 0.0f;
  if (rho>255.0f) rho = 255.0f;

  dp_bmap[ithr] = (unsigned char) rho;
}

void makebmap_cuda(int NBLK, int NTHR, float *dp_map, int NSIZE, float *dp_tmp, unsigned char *dp_bmap)
{
  makebmap_kern<<<NBLK, NTHR>>>((float*)dp_map, NSIZE, (float*)dp_tmp, (unsigned char *)dp_bmap);
}


